#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void upsweep(int twod, int offset, int* output)
{
	int index = threadIdx.x;

	if (index < twod) {
		int ai = offset * (2 * index + 1) - 1;
		int bi = offset * (2 * index + 2) - 1;

		output[bi] += output[ai];

	}
}


__global__ void downsweep(int twod, int offset, int* output)
{
	int index = threadIdx.x;

	if (index < twod) {
		int ai = offset * (2 * index + 1) - 1;
		int bi = offset * (2 * index + 2) - 1;

		int t = output[ai];
		output[ai] = output[bi];
		output[bi] += t;

	}
}


__global__ void pairs_repeat(int n, int* x, int* x_shift, int* repeat)
{
	int index = threadIdx.x;

	if (index > 0)
		repeat[index-1] = x[index] == x_shift[index];


}

extern void use_upsweep(int twod, int offset, int* output)
{
	int threadsPerBlock = 1024;
	//int numberOfBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

	upsweep << <1, threadsPerBlock >> > (twod, offset, output);
	hipDeviceSynchronize();
}

extern void use_downsweep(int twod, int offset, int* output)
{
	int threadsPerBlock = 1024;
	//int numberOfBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

	downsweep << <1, threadsPerBlock >> > (twod, offset, output);
	hipDeviceSynchronize();
}

extern void use_pairs_repeat(int n, int* x, int* x_shift, int* repeat)
{
	int threadsPerBlock = 1024;
	//int numberOfBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

	pairs_repeat << <1, threadsPerBlock >> > (n, x, x_shift, repeat);
	hipDeviceSynchronize();
}
