#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

using namespace std;

__global__ void upsweep(int twod, int offset, int* output)
{
	int index = threadIdx.x;
	//int stride = blockDim.x * gridDim.x;
	//int twod1 = twod * 2;

	if (index < twod) {
		int ai = offset * (2 * index + 1) - 1;
		int bi = offset * (2 * index + 2) - 1;

		output[bi] += output[ai];

	}
}


__global__ void downsweep(int twod, int offset, int* output)
{
	int index = threadIdx.x;
	//int stride = blockDim.x * gridDim.x;
	//int twod1 = twod * 2;

	if (index < twod) {
		int ai = offset * (2 * index + 1) - 1;
		int bi = offset * (2 * index + 2) - 1;

		int t = output[ai];
		output[ai] = output[bi];
		output[bi] += t;

	}
}

extern void use_upsweep(int twod, int offset, int* output)
{
	int threadsPerBlock = 1024;
	//int numberOfBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

	upsweep << <1, threadsPerBlock >> > (twod, offset, output);
	hipDeviceSynchronize();
}

extern void use_downsweep(int twod, int offset, int* output)
{
	int threadsPerBlock = 1024;
	//int numberOfBlocks = (N + threadsPerBlock - 1) / threadsPerBlock;

	downsweep << <1, threadsPerBlock >> > (twod, offset, output);
	hipDeviceSynchronize();
}